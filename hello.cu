#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include "timer.h"

__global__ void vector_add(float *a,float *b, float *c, int N){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i < N){
        c[i] = a[i] + b[i];
    }
}


int main(){

    hipDeviceSynchronize();

    const int N = 1000000;


    float *a = new float[N];
    float *b = new float[N];
    float *out = new float[N];
    float *out_p = new float[N];

    for (int i = 0; i < N; i++){
        a[i] = rand();
        b[i] = rand();
    }

    const size_t Size = N * sizeof(float);

    core::timer cpu_t;
    cpu_t.start();
    for (int i = 0; i < N; i++){
        out[i] = a[i] + b[i];
    }
    printf("Cpu time taken :- %f ns\n",cpu_t.nanoseconds());

    // printf("array from the cpu \n");
    // for (int i = 0; i < N; i++){
    //     printf("%d ",out[i]);
    // }

    core::timer gpu_total_t;
    gpu_total_t.start();

    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, Size);
    hipMalloc(&d_b, Size);
    hipMalloc(&d_out, Size);

    hipMemcpy(d_a,a,Size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,Size,hipMemcpyHostToDevice);

    //still don't understand this part :(
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    core::timer gpu_t;
    gpu_t.start();

    vector_add<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_out,N);

    printf("gpu time taken :- %f ns\n",gpu_t.nanoseconds());

    hipMemcpy(out_p,d_out,Size,hipMemcpyDeviceToHost);

    // printf("array from the gpu \n");
    // for (int i = 0; i < N; i++){
    //     printf("%d ",out_p[i]);
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    printf("gpu time taken :- %f ns\n",gpu_total_t.nanoseconds());

    return 0;
}