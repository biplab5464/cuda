#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include<stdio.h>
#include "timer.h"

__global__ void matrix_add(float *a,float *b, float *c, int N,int M){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if(i < N && i < M){
        c[ i * N + j ] = a[i * N + j] + b[i * N + j];
    }
}


int main(){

    hipDeviceSynchronize();

    const int N = 10000;
    const int M = 10000;


    //float a[N][M], b[N][M], out[N][M], out_p[N][M];
    float *a = (float*)malloc(N * M * sizeof(float));
    float *b = (float*)malloc(N * M * sizeof(float));
    float *out = (float*)malloc(N * M * sizeof(float));
    float *out_p = (float*)malloc(N * M * sizeof(float));

    for (int i = 0; i < N; i++){
        for (int j = 0; j < M; j++){
            a[i * N + j] = rand() / (float)RAND_MAX;
            b[i * N + j] = rand() / (float)RAND_MAX;
        }
    }

    const size_t Size = N * M * sizeof(float);

    core::timer cpu_t;
    cpu_t.start();
    for (int i = 0; i < N; i++){
        for(int j = 0; j < M;j++){
            out[i * N + j] = a[i * N + j] + b[i * N + j];
        }
    }
    printf("Cpu time taken :- %f sec\n", cpu_t.nanoseconds()/1000000000);

    // printf("added matrix from cpu\n");
    // for (int i = 0; i < N; i++){
    //     for(int j = 0; j < M;j++){
    //         printf("%f ",out[i * N + j]);
    //     }
    //     printf("\n");
    // }

    core::timer gpu_total_t;
    gpu_total_t.start();

    float *d_a, *d_b, *d_out;
    hipMalloc(&d_a, Size);
    hipMalloc(&d_b, Size);
    hipMalloc(&d_out, Size);

    hipMemcpy(d_a,a,Size,hipMemcpyHostToDevice);
    hipMemcpy(d_b,b,Size,hipMemcpyHostToDevice);

    //still don't understand this part :(
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, (M + threadsPerBlock.y - 1) / threadsPerBlock.y);

    core::timer gpu_t;
    gpu_t.start();

    matrix_add<<<blocksPerGrid,threadsPerBlock>>>(d_a,d_b,d_out,N,M);

   printf("gpu time taken :- %f sec\n", gpu_t.nanoseconds()/1000000000);

    hipMemcpy(out_p,d_out,Size,hipMemcpyDeviceToHost);

    // printf("added matrix from gpu\n");
    // for (int i = 0; i < N; i++){
    //     for(int j = 0; j < M;j++){
    //         printf("%f ",out_p[i * N + j]);
    //     }
    //     printf("\n");
    // }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    printf("gpu total time taken :- %f sec\n",gpu_total_t.nanoseconds()/1000000000);
    
    return 0;
}